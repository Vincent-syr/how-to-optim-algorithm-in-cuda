
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include <iostream>
#include <vector>

using namespace std;

#include<stdio.h>
#include<time.h>
#define WARPSIZE 32
__global__ void kernel1(float* A) {
    __shared__ float data[32][32];
    int tid = threadIdx.x;
    int col = tid/WARPSIZE;
    int row = tid%WARPSIZE;
    data[row][col] = 100.f;
    A[tid] = data[row][col];
}


__global__ void kernel2(float* A) {
    __shared__ float data[32][32];
    int tid = threadIdx.x;
    int row = tid/WARPSIZE;
    int col = tid%WARPSIZE;
    data[row][col] = 100.f;
    A[tid] = data[row][col];
}

int main() {
    int blocksize = 32*32;
    float* h_A = (float*)malloc(sizeof(float)*blocksize);
    float* d_A;
    hipMalloc(&d_A, sizeof(float)*blocksize);
 
    kernel1<<<1, blocksize>>>(d_A);
    hipDeviceSynchronize();
    hipMemcpy(h_A, d_A, blocksize*sizeof(float), hipMemcpyDeviceToHost);

    kernel2<<<1, blocksize>>>(d_A);
    hipDeviceSynchronize();
    hipMemcpy(h_A, d_A, blocksize*sizeof(float), hipMemcpyDeviceToHost);    

    hipFree(d_A);
    free(h_A);
    return 0;
}
